#include "hip/hip_runtime.h"
#include "utility_header.cuh"

#include "camera.cuh"
#include "hittable.cuh"
#include "hittable_list.cuh"
#include "material.cuh"
#include "sphere.cuh"


__global__ void create_world(hiprandState *rand_state,hittable **d_list, hittable **d_world,int *d_count) {

    hiprandState local_rand_state = *rand_state;
   
    //Allocate memory for the hittable list
    d_list[0] = nullptr; //First element is a dummy pointer

    for(int i = 1; i < 22*22+1+3; i++) {
        d_list[i] = nullptr;
    }
   if (threadIdx.x == 0 && blockIdx.x == 0) {
        *d_count = 1;
        material *ground_material = new lambertian(color(0.5f,0.5f,0.5f));
        d_list[0] = new sphere(vec3(0,-1000.0f,-1), 1000, ground_material);


        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = random_float(&local_rand_state);
                point3 center(a + 0.9f*random_float(&local_rand_state),
                             0.2f,
                             b+0.9f*random_float(&local_rand_state));
                
                if ((center - point3(4,0.2f,0)).length() > 0.9f){
                    if(choose_mat < 0.8f) { //Lambertian
                        color albedo = color::random(&local_rand_state) * color::random(&local_rand_state);
                        point3 center2 = center + vec3(0, random_float(&local_rand_state,0,0.5f), 0);
                        d_list[(*d_count)++] = new sphere(center, center2, 0.2F, new lambertian(albedo));
                    }
                    else if(choose_mat < 0.95f) { //Metal
                        color albedo = color::random(&local_rand_state, 0.5f, 1.0f);
                        float fuzz = random_float(&local_rand_state) * 0.5f;
                        d_list[(*d_count)++] = new sphere(center, 0.2f,new metal(albedo, fuzz));
                    
                    }
                    else { //Dielectric
                        d_list[(*d_count)++] = new sphere(center, 0.2, new dielectric(1.5));
                    }
                }
            }
        }
        d_list[(*d_count)++] = new sphere(vec3(0, 1,0), 1.0f, new dielectric(1.5));
        d_list[(*d_count)++] = new sphere(vec3(-4, 1, 0), 1.0f, new lambertian(vec3(0.4f, 0.2f, 0.1f)));
        d_list[(*d_count)++] = new sphere(vec3(4, 1, 0),  1.0f, new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list,*d_count);
    }

}

__global__ void free_world(hittable **d_list, hittable **d_world,int num_hittables) {
     for (int i = 0; i < num_hittables; i++) {
        if(d_list[i]){
            delete ((sphere *)d_list[i])->mat_ptr;
            delete d_list[i];
        }
    }
    delete *d_world;
}

__host__ int main(){
    hipError_t err = hipSetDevice(0);
if (err != hipSuccess) {
    std::cerr << "hipSetDevice failed: " << hipGetErrorString(err) << std::endl;
    return 1;
}

    //Create camera
    camera cam;
    initialize_camera(
        cam,
        16.0f/9.0f, //Aspect ratio
        1200, //Image width
        100, //Samples per pixel
        50, //Max depth
        20.0f, //Field of view in degrees
        point3(13,2,3), //Look from
        point3(0,0,0), //Look at
        vec3(0,1,0), //Up vector
        0.0f, //Defocus angle
        10.0f //Focus distance
    );


    //Allocate Camera Data
    camera *d_cam;
    checkCudaErrors(hipMalloc((void**)&d_cam, sizeof(camera)));
    checkCudaErrors(hipMemcpy(d_cam, &cam, sizeof(camera), hipMemcpyHostToDevice));

    //CUDA Image Division
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << cam.image_width << "x" << cam.image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = cam.image_width*cam.image_height;
    size_t fb_size = 3*num_pixels*sizeof(float); //frame buffer

    //Allocate frame buffer
    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb , fb_size));


    //Set up CUDA Random state
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState))); //Random state for each pixel

    //Initialise Render

    dim3 blocks(cam.image_width/tx+1,cam.image_height/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(d_rand_state,d_cam); //Initialize the random state and camera
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    //Make World of hittables
    hittable **d_list;
    int max_hittables = 22*22+1+3; //22x22 spheres + 4 other spheres
    checkCudaErrors(hipMalloc((void **) &d_list, max_hittables*sizeof(hittable *)));

    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable *)));
    int *d_count;
    checkCudaErrors(hipMallocManaged(&d_count, sizeof(int)));
    create_world<<<1,1>>>(d_rand_state,d_list,d_world,d_count);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

 

    render<<<blocks, threads>>>(d_rand_state,fb,d_world,d_cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Write image
    std::cout << "P3\n" << cam.image_width << " " << cam.image_height << "\n255\n";
    for (int j = cam.image_height-1; j >= 0; j--) {
        for (int i = 0; i < cam.image_width; i++) {
            size_t pixel_index = j*cam.image_width + i;
            write_color(std::cout,fb[pixel_index]);
        }
    }

    //Cleanup
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,*d_count);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_cam));

    
    hipDeviceReset();
}




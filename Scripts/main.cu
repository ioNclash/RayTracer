#include "hip/hip_runtime.h"
#include "utility_header.cuh"

#include "camera.cuh"
#include "hittable.cuh"
#include "hittable_list.cuh"
#include "sphere.cuh"


__global__ void create_world(hittable **d_list, hittable **d_world ){
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5f);
        *(d_list+1) = new sphere(vec3(0,-100.5f,-1), 100);
        *(d_list+2) = new sphere(vec3(1,0,-1), 0.5f);
        *(d_list+3) = new sphere(vec3(-1,0,-1), 0.5f);
        *(d_list+4) = new sphere(vec3(0,1,-1), 0.5f);

        *d_world    = new hittable_list(d_list,5);
    }

}

__global__ void free_world(hittable **d_list, hittable **d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}


__host__ int main(){
    //Create camera
    camera cam;

    cam.set_aspect_ratio(16.0f/9.0f);
    cam.set_image_width(1600);

    //Allocate Camera Data
    camera *d_cam;
    checkCudaErrors(hipMallocManaged((void**)&d_cam, sizeof(cam)));
    *d_cam = cam;

    //CUDA Image Division
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << d_cam->get_image_width() << "x" << d_cam->get_image_height() << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = d_cam->get_image_width()*d_cam->get_image_height();
    size_t fb_size = 3*num_pixels*sizeof(float); //frame buffer

    //Allocate frame buffer
    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb , fb_size));



    //Make World of hittables
    hittable **d_list;
    checkCudaErrors(hipMalloc((void **) &d_list, 2*sizeof(hittable *)));

    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable *)));
    create_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Set up CUDA Random state
    hiprandState * d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState))); //Random state for each pixel

    //Render buffer

    dim3 blocks(d_cam->get_image_width()/tx+1,d_cam->get_image_height()/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(d_cam, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render<<<blocks, threads>>>(fb,d_world,d_cam,d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Write image
    std::cout << "P3\n" << d_cam->get_image_width() << " " << d_cam->get_image_height() << "\n255\n";
    for (int j = d_cam->get_image_height()-1; j >= 0; j--) {
        for (int i = 0; i < d_cam->get_image_width(); i++) {
            size_t pixel_index = j*d_cam->get_image_width() + i;
            write_color(std::cout,fb[pixel_index]);
        }
    }

    //Cleanup
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_cam));

    
    hipDeviceReset();
}




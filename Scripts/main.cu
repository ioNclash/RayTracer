#include "hip/hip_runtime.h"
#include "color.cuh"
#include "ray.cuh"
#include "vec3.cuh"

#include <iostream>

__device__ color ray_color(const ray&r) {
    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0f, 1.0f, 1.0f) + a*color(0.5f, 0.7f, 1.0f);
}

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
    if(result){
        std::cerr << "CUDA error =" << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";

        //Ensure device reset on exit
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(color *fb, int max_x, int max_y,
    point3 bottom_left_pixel, vec3  pixel_width, vec3 pixel_height, point3 camera_center){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return; //Due to block size, prevent rendering outside of image

    point3 pixel_center = bottom_left_pixel + (i*pixel_width) + (j*pixel_height);
    vec3 ray_direction = pixel_center - camera_center;
    ray r(camera_center,ray_direction);
    int pixel_index = j*max_x + i; //get index in frame buffer
    fb[pixel_index] = ray_color(r);
}


__host__ int main(){

    //Image
    float aspect_ratio= 16.0f/9.0f;
    int image_width = 1600;  // image width
    int image_height = int(image_width/aspect_ratio);
    image_height = (image_height <1 ) ? 1 : image_height;   // image height

    //Camera
    float focal_length = 1.0f;
    float viewport_height = 2.0f;
    float viewport_width = viewport_height * (float(image_width)/float(image_height));
    point3 camera_center = point3(0,0,0);

    //Calculate vectors across viewport edges
    vec3 viewport_u = vec3(viewport_width,0,0);
    vec3 viewport_v = vec3(0,viewport_height,0); //+ here to make pixel00 the bottom left pixel

    vec3 pixel_delta_u = viewport_u/image_width;
    vec3 pixel_delta_v = viewport_v/image_height;

    point3 viewport_bottom_left = camera_center - vec3(0,0,focal_length) - viewport_u/2 - viewport_v/2;
    point3 pixel00_loc = viewport_bottom_left + 0.5f * (pixel_delta_u + pixel_delta_v);


    //CUDA Image Division
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = image_width*image_height;
    size_t fb_size = 3*num_pixels*sizeof(float); //frame buffer

    //Allocate frame buffer
    color *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb , fb_size));


    //Render buffer

    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb,image_width,image_height,pixel00_loc,pixel_delta_u,pixel_delta_v,camera_center);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Write image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height-1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j*image_width + i;
            write_color(std::cout,fb[pixel_index]);
        }
    }
    checkCudaErrors(hipFree(fb));
}




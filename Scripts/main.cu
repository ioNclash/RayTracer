
#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
    if(result){
        std::cerr << "CUDA error =" << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";

        //Ensure device reset on exit
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return; //Due to block size, prevent rendering outside of image
    int pixel_index = j*max_x*3 + i*3; //get index in frame buffer
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}


__host__ int main(){

    int nx = 1600;  // image width
    int ny = 900;   // image height
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float); //frame buffer

    //Allocate framb buffer
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb , fb_size));


    //Render buffer

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx,ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //Write image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*3*nx + i*3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(fb));
}



